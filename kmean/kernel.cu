#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
#include<math.h>
#include<float.h>
#include "hip/hip_runtime.h"


#define MAX_NUM_THREADS 1024

__device__ int *resultIndexCluster;
__device__ Point *clusters, *points;

/*
* Finds the index of the closest cluster to point.
* Return an array of cluster indexes
*/
__global__ void minDistanceCluster(Point *clusters, Point *points, int* resultIndexCluster, GlobalVar data)
{
	double pX, pY, cX, cY;
	double currentD, minD = DBL_MAX;
	int pIndex;
	pIndex = blockIdx.x * MAX_NUM_THREADS + threadIdx.x;
	
	if (pIndex < data.sizePointsArr)
	{
		pX = points[pIndex].x;
		pY = points[pIndex].y;
		for (int i = 0; i < data.sizeClusterArr; i++)
		{
			cX = clusters[i].x;
			cY = clusters[i].y;
			currentD = (cX - pX) * (cX - pX) + (cY - pY) * (cY - pY);
			if (currentD < minD){
				minD = currentD;
				resultIndexCluster[pIndex] = i;
			}
		}
	}
}

/*
* memcopy clusters and call kernel
*/
void closestClusterToPoint(int* indexMinCluster, Point* clusterArr, GlobalVar data)
{
	int i;
	int numBlock = (int)ceil((double)data.sizePointsArr / MAX_NUM_THREADS);

	hipError_t cudaStatus;
	dim3 dimGrid(numBlock);
	dim3 dimBlock(MAX_NUM_THREADS);

	//Copy cluster array to device:
	cudaStatus = hipMemcpy(clusters, clusterArr, data.sizeClusterArr * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "hipMemcpy (clusters) failed!\n");
		fflush(stdout);
		return;
	}

	minDistanceCluster <<<dimGrid, dimBlock >>>(clusters, points, resultIndexCluster, data);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "distanceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		fflush(stdout);
		return;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		fflush(stdout);
		return;
	}

	//Read result from device:
	hipMemcpy(indexMinCluster, resultIndexCluster, data.sizePointsArr*sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "hipMemcpy failed!\n");
		fflush(stdout);
		return;
	}
}

/*
* Malloc and memcopy only once per iteration
*/
void prepForCuda(GlobalVar data)
{
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.				///------------- should be according to device id?? --------------
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		fflush(stdout);
		return;
	}

	//allocate memory in device for cluster array:
	cudaStatus = hipMalloc(&clusters, data.sizeClusterArr *sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "hipMalloc (clusters) failed!\n");
		fflush(stdout);
		return;
	}

	//allocate memory in device for points array:
	cudaStatus = hipMalloc(&points, data.sizePointsArr *sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "hipMalloc (points) failed!\n");
		fflush(stdout);
		return;
	}

	//allocate memory in device for result:
	cudaStatus = hipMalloc(&resultIndexCluster, data.sizePointsArr *sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "hipMalloc failed!\n");
		fflush(stdout);
		return;
	}
}

//free memory from device:
void freeAllocationCuda()
{
	hipFree(resultIndexCluster);
	hipFree(clusters);
	hipFree(points);
}

void copyPoints(Point* pointsArr, GlobalVar data)
{
	hipError_t cudaStatus;

	//Copy points array to device:
	cudaStatus = hipMemcpy(points, pointsArr, data.sizePointsArr * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "hipMemcpy (points) failed!\n");
		fflush(stdout);
		return;
	}
}